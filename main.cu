#include "common.h"
#include "mmio.h"
#include "test.h"

float clockElapsed(hipEvent_t evt_start, hipEvent_t evt_stop) {
    hipEventSynchronize(evt_stop);

    float elapsedTime = 0;

    hipEventElapsedTime(&elapsedTime, evt_start, evt_stop);
    elapsedTime *= 1000;  // Returns in microseconds

    return elapsedTime;
}

int main(int argc, char** argv) {
    // report precision of floating-point
    printf("---------------------------------------------------------------------------------------------\n");
    char* precision;
    if (sizeof(VALUE_TYPE) == 4) {
        precision = (char*)"32-bit Single Precision";
    } else if (sizeof(VALUE_TYPE) == 8) {
        precision = (char*)"64-bit Double Precision";
    } else {
        printf("Wrong precision. Program exit!\n");
        return 0;
    }

    printf("PRECISION = %s\n", precision);
    printf("Benchmark REPEAT = %i\n", BENCH_REPEAT);
    printf("---------------------------------------------------------------------------------------------\n");

    int m, n, nnzA;
    int* csrRowPtrA;
    int* csrColIdxA;
    VALUE_TYPE* csrValA;

    // ex: ./spmv webbase-1M.mtx
    int argi = 1;

    char* filename;
    if (argc > argi) {
        filename = argv[argi];
        argi++;
    }

    printf("-------------- %s --------------\n", filename);

    int device_id = 0;

    if (argc > argi) {
        device_id = atoi(argv[2]);
        argi++;
    }

    CUDA_CHK(hipSetDevice(device_id));

    int wpb = WARP_PER_BLOCK;
    if (argc > argi) {
        wpb = atoi(argv[3]);
        argi++;
    }

    printf("WARPS PER BLOCK = %i.\n", wpb);

    // read matrix from mtx file
    int ret_code;
    MM_typecode matcode;
    FILE* f;

    int nnzA_mtx_report;
    int isInteger = 0, isReal = 0, isPattern = 0, isSymmetric = 0;

    // load matrix
    if ((f = fopen(filename, "r")) == NULL)
        return -1;

    if (mm_read_banner(f, &matcode) != 0) {
        printf("Could not process Matrix Market banner.\n");
        return -2;
    }

    if (mm_is_complex(matcode)) {
        printf("Sorry, data type 'COMPLEX' is not supported.\n");
        return -3;
    }

    char *pch, *pch1;
    pch = strtok(filename, "/");
    while (pch != NULL) {
        pch1 = pch;
        pch = strtok(NULL, "/");
    }

    pch = strtok(pch1, ".");

    if (mm_is_pattern(matcode)) {
        isPattern = 1;
    }
    if (mm_is_real(matcode)) {
        isReal = 1;
    }
    if (mm_is_integer(matcode)) {
        isInteger = 1;
    }

    /* find out size of sparse matrix .... */
    ret_code = mm_read_mtx_crd_size(f, &m, &n, &nnzA_mtx_report);
    if (ret_code != 0)
        return -4;

    if (n != m) {
        printf("Matrix is not square.\n");
        return -5;
    }

    if (mm_is_symmetric(matcode) || mm_is_hermitian(matcode)) {
        isSymmetric = 1;
        printf("input matrix is symmetric = true\n");
    } else {
        printf("input matrix is symmetric = false\n");
    }

    int* csrRowPtrA_counter = (int*)malloc((m + 1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m + 1) * sizeof(int));

    int* csrRowIdxA_tmp = (int*)malloc(nnzA_mtx_report * sizeof(int));
    int* csrColIdxA_tmp = (int*)malloc(nnzA_mtx_report * sizeof(int));
    VALUE_TYPE* csrValA_tmp = (VALUE_TYPE*)malloc(nnzA_mtx_report * sizeof(VALUE_TYPE));

    /* NOTE: when reading in doubles, ANSI C requires the use of the "l"  */
    /*   specifier as in "%lg", "%lf", "%le", otherwise errors will occur */
    /*  (ANSI C X3.159-1989, Sec. 4.9.6.2, p. 136 lines 13-15)            */

    for (int i = 0; i < nnzA_mtx_report; i++) {
        int idxi, idxj;
        double fval;
        int ival;
        int returnvalue;

        if (isReal)
            returnvalue = fscanf(f, "%d %d %lg\n", &idxi, &idxj, &fval);
        else if (isInteger) {
            returnvalue = fscanf(f, "%d %d %d\n", &idxi, &idxj, &ival);
            fval = ival;
        } else if (isPattern) {
            returnvalue = fscanf(f, "%d %d\n", &idxi, &idxj);
            fval = 1.0;
        }

        // adjust from 1-based to 0-based
        idxi--;
        idxj--;

        csrRowPtrA_counter[idxi]++;
        csrRowIdxA_tmp[i] = idxi;
        csrColIdxA_tmp[i] = idxj;
        csrValA_tmp[i] = fval;
    }

    if (f != stdin)
        fclose(f);

    if (isSymmetric) {
        for (int i = 0; i < nnzA_mtx_report; i++) {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i])
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
        }
    }

    // exclusive scan for csrRowPtrA_counter
    int old_val, new_val;

    old_val = csrRowPtrA_counter[0];
    csrRowPtrA_counter[0] = 0;
    for (int i = 1; i <= m; i++) {
        new_val = csrRowPtrA_counter[i];
        csrRowPtrA_counter[i] = old_val + csrRowPtrA_counter[i - 1];
        old_val = new_val;
    }

    nnzA = csrRowPtrA_counter[m];
    csrRowPtrA = (int*)malloc((m + 1) * sizeof(int));
    memcpy(csrRowPtrA, csrRowPtrA_counter, (m + 1) * sizeof(int));
    memset(csrRowPtrA_counter, 0, (m + 1) * sizeof(int));

    csrColIdxA = (int*)malloc(nnzA * sizeof(int));
    csrValA = (VALUE_TYPE*)malloc(nnzA * sizeof(VALUE_TYPE));

    if (isSymmetric) {
        for (int i = 0; i < nnzA_mtx_report; i++) {
            if (csrRowIdxA_tmp[i] != csrColIdxA_tmp[i]) {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;

                offset = csrRowPtrA[csrColIdxA_tmp[i]] + csrRowPtrA_counter[csrColIdxA_tmp[i]];
                csrColIdxA[offset] = csrRowIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrColIdxA_tmp[i]]++;
            } else {
                int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
                csrColIdxA[offset] = csrColIdxA_tmp[i];
                csrValA[offset] = csrValA_tmp[i];
                csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
            }
        }
    } else {
        for (int i = 0; i < nnzA_mtx_report; i++) {
            int offset = csrRowPtrA[csrRowIdxA_tmp[i]] + csrRowPtrA_counter[csrRowIdxA_tmp[i]];
            csrColIdxA[offset] = csrColIdxA_tmp[i];
            csrValA[offset] = csrValA_tmp[i];
            csrRowPtrA_counter[csrRowIdxA_tmp[i]]++;
        }
    }

    printf("input matrix A: ( %i, %i ) nnz = %i\n", m, n, nnzA);

    // extract L with the unit-lower triangular sparsity structure of A
    int nnzL = 0;
    int* csrRowPtrL_tmp = (int*)malloc((m + 1) * sizeof(int));
    int* csrColIdxL_tmp = (int*)malloc(nnzA * sizeof(int));
    VALUE_TYPE* csrValL_tmp = (VALUE_TYPE*)malloc(nnzA * sizeof(VALUE_TYPE));

    int nnz_pointer = 0;
    csrRowPtrL_tmp[0] = 0;
    for (int i = 0; i < m; i++) {
        for (int j = csrRowPtrA[i]; j < csrRowPtrA[i + 1]; j++) {
            if (csrColIdxA[j] < i) {
                csrColIdxL_tmp[nnz_pointer] = csrColIdxA[j];
                csrValL_tmp[nnz_pointer] = 1.0;  // csrValA[j];
                nnz_pointer++;
            } else {
                break;
            }
        }

        csrColIdxL_tmp[nnz_pointer] = i;
        csrValL_tmp[nnz_pointer] = 1.0;
        nnz_pointer++;

        csrRowPtrL_tmp[i + 1] = nnz_pointer;
    }

    nnzL = csrRowPtrL_tmp[m];
    printf("A's unit-lower triangular L: ( %i, %i ) nnz = %i\n", m, n, nnzL);

    csrColIdxL_tmp = (int*)realloc(csrColIdxL_tmp, sizeof(int) * nnzL);
    csrValL_tmp = (VALUE_TYPE*)realloc(csrValL_tmp, sizeof(VALUE_TYPE) * nnzL);

    // run serial syncfree SpTS as a reference
    printf("---------------------------------------------------------------------------------------------\n");

    // set device
    hipSetDevice(device_id);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device_id);

    printf("---------------------------------------------------------------------------------------------\n");
    printf("Device [ %i ] %s @ %4.2f MHz\n", device_id, deviceProp.name, deviceProp.clockRate * 1e-3f);

    test_solve_L_analysis_multirow(pch, csrRowPtrL_tmp, csrColIdxL_tmp, csrValL_tmp, n);

    printf("Bye!\n");

    // done!
    free(csrColIdxA);
    free(csrValA);
    free(csrRowPtrA);

    free(csrColIdxL_tmp);
    free(csrValL_tmp);
    free(csrRowPtrL_tmp);

    return 0;
}
